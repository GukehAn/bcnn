
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2016-present Jean-Noel Braun.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#ifdef BCNN_USE_CUDA

#include "bcnn_maxpool_layer.h"
#include "bcnn_utils.h"

__global__ void bcnn_forward_maxpool_layer_kernel(int n, int in_h, int in_w,
                                                  int in_c, int stride,
                                                  int size, float *input,
                                                  float *output, int *indexes) {
    int h = (in_h - 1) / stride + 1;
    int w = (in_w - 1) / stride + 1;
    int c = in_c;

    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) {
        return;
    }

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int out_index = j + w * (i + h * (k + c * b));
    float max = -INFINITY;
    int max_i = -1;
    int l, m;
    for (l = 0; l < size; ++l) {
        for (m = 0; m < size; ++m) {
            int cur_h = i * stride + l;
            int cur_w = j * stride + m;
            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            int valid =
                (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w);
            float val = (valid != 0) ? input[index] : -INFINITY;
            max_i = (val > max) ? index : max_i;
            max = (val > max) ? val : max;
        }
    }
    output[out_index] = max;
    indexes[out_index] = max_i;
}

void bcnn_forward_maxpool_layer_gpu(bcnn_net *net, bcnn_node *node) {
    bcnn_tensor *src_tensor = &net->tensors[node->src[0]];
    bcnn_tensor *dst_tensor = &net->tensors[node->dst[0]];
    bcnn_maxpool_param *param = (bcnn_maxpool_param *)node->param;
#ifdef BCNN_USE_CUDNN
    float zero = 0.0f, one = 1.0f;
    bcnn_cudnn_check(
        cudnnPoolingForward(bcnn_cudnn_handle(), param->pooling_desc, &one,
                            param->src_tensor_desc, src_tensor->data_gpu, &zero,
                            param->dst_tensor_desc, dst_tensor->data_gpu));
#else
    int sz = bcnn_tensor_size(dst_tensor);

    bcnn_forward_maxpool_layer_kernel<<<bcnn_cuda_gridsize(sz),
                                        BCNN_CUDA_THREADS>>>(
        sz, src_tensor->h, src_tensor->w, src_tensor->c, param->stride,
        param->size, src_tensor->data_gpu, dst_tensor->data_gpu,
        param->indexes_gpu);
    bcnn_cuda_check(cudaPeekAtLastError());
#endif

    return;
}

__global__ void bcnn_backward_maxpool_layer_kernel(int n, int in_h, int in_w,
                                                   int in_c, int stride,
                                                   int size, float *diff,
                                                   float *prev_delta,
                                                   int *indexes) {
    int h = (in_h - 1) / stride + 1;
    int w = (in_w - 1) / stride + 1;
    int c = in_c;
    int area = (size - 1) / stride;

    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) {
        return;
    }

    int index = id;
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    int w_offset = (-size - 1) / 2 + 1;
    int h_offset = (-size - 1) / 2 + 1;

    float d = 0;
    int l, m;
    for (l = -area; l < area + 1; ++l) {
        for (m = -area; m < area + 1; ++m) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;
            int out_index = out_w + w * (out_h + h * (k + c * b));
            int valid = (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h);
            d += (valid && indexes[out_index] == index) ? diff[out_index] : 0;
        }
    }
    prev_delta[index] += d;
}

void bcnn_backward_maxpool_layer_gpu(bcnn_net *net, bcnn_node *node) {
    bcnn_tensor *src_tensor = &net->tensors[node->src[0]];
    bcnn_tensor *dst_tensor = &net->tensors[node->dst[0]];
    bcnn_maxpool_param *param = (bcnn_maxpool_param *)node->param;
#ifdef BCNN_USE_CUDNN
    float zero = 0.0f, one = 1.0f;
    bcnn_cudnn_check(cudnnPoolingBackward(
        bcnn_cudnn_handle(), param->pooling_desc, &one, param->dst_tensor_desc,
        dst_tensor->data_gpu, param->dst_tensor_desc, dst_tensor->grad_data_gpu,
        param->src_tensor_desc, src_tensor->data_gpu, &zero,
        param->src_tensor_desc, src_tensor->grad_data_gpu));
#else
    int sz = bcnn_tensor_size(src_tensor);

    bcnn_backward_maxpool_layer_kernel<<<bcnn_cuda_gridsize(sz),
                                         BCNN_CUDA_THREADS>>>(
        sz, src_tensor->h, src_tensor->w, src_tensor->c, param->stride,
        param->size, dst_tensor->grad_data_gpu, src_tensor->grad_data_gpu,
        param->indexes_gpu);
    bcnn_cuda_check(cudaPeekAtLastError());
#endif

    return;
}

#endif