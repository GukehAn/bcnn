
#include <hip/hip_runtime.h>
/*
* Copyright (c) 2016 Jean-Noel Braun.
*
* Permission is hereby granted, free of charge, to any person obtaining a copy
* of this software and associated documentation files (the "Software"), to deal
* in the Software without restriction, including without limitation the rights
* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
* copies of the Software, and to permit persons to whom the Software is
* furnished to do so, subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in
* all copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
* SOFTWARE.
*/


#ifdef BCNN_USE_CUDA

#include <bh/bh.h>

#include "bcnn/bcnn.h"

__global__ void _bcnn_forward_activation_layer_kernel(float *x, int sz, bcnn_activation a)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < sz) {
		switch (a) {
		case TANH:
			x[i] = (exp(2 * x[i]) - 1) / (exp(2 * x[i]) + 1);
			break;
		case RELU:
			x[i] = x[i] * (x[i] > 0);
			break;
		case RAMP:
			x[i] = x[i] * (x[i] > 0) + 0.1 * x[i];
			break;
		case CLAMP:
			x[i] = bh_clamp(x[i], 0, 1);
			break;
		}
	}
	return;
}

int bcnn_forward_activation_gpu(float *x, int sz, bcnn_activation a)
{
	_bcnn_forward_activation_layer_kernel<<<bcnn_cuda_gridsize(sz), BCNN_CUDA_THREADS>>>(x,
		sz, layer->activation);
	return BCNN_SUCCESS;
}

int bcnn_forward_activation_layer_gpu(bcnn_layer *layer, bcnn_workload *wrk)
{
	int sz = layer->output_shape[0] * layer->output_shape[1] * layer->output_shape[2] *
		wrk->batch_size;

	layer->output_gpu = wrk->input_gpu;
	bcnn_forward_activation_gpu(layer->output_gpu, sz, layer->activation);
	bcnn_cuda_check(cudaPeekAtLastError());

	return BCNN_SUCCESS;
}


__global__ void _bcnn_backward_activation_layer_kernel(float *x, float *diff, int sz, bcnn_activation a)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < sz) {
		switch (a) {
		case TANH:
			diff[i] *= (1 - x[i] * x[i]);
			break;
		case RELU:
			diff[i] *= ((float)(x[i] > 0));
			break;
		case RAMP:
			diff[i] *= ((float)(x[i] > 0) + 0.1f);
			break;
		case CLAMP:
			diff[i] *= (float)(x[i] > 0.0f && (x[i] < 1.0f));
			break;
		}
	}
}

int bcnn_backward_activation_gpu(float *x, float *dx, int sz, bcnn_activation a)
{
	_bcnn_backward_activation_layer_kernel<<<bcnn_cuda_gridsize(sz), BCNN_CUDA_THREADS>>>(x, dx
		sz, layer->activation);
	return BCNN_SUCCESS;
}

int bcnn_backward_activation_layer_gpu(bcnn_layer *layer, bcnn_workload *wrk)
{
	int sz = layer->output_shape[0] * layer->output_shape[1] * layer->output_shape[2] *
		wrk->batch_size;
	
    bcnn_backward_activation_gpu(layer->output, layer->diff, sz, layer->activation)
	bcnn_cuda_check(cudaPeekAtLastError());
	wrk->diff = layer->diff_gpu;

	return BCNN_SUCCESS;
}


#endif